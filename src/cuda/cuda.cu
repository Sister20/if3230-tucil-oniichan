#include <stdio.h>
#include <iostream>
#include "inverse-matrix.cuh"

using namespace std;

// Display a 2d matrix
void printMatrix(double *matrix, int count_width, int count_row, int res)
{
  for (int i = 0; i < count_row; i++)
  {
    for (int j = 0; j < count_width; j++)
    {
      if (res)
      {
        if (j > (count_width / 2) - 1)
        {
          cout << *(matrix + j + i * count_width) << " ";
        }
      }
      else
      {
        cout << *(matrix + j + i * count_width) << " ";
        if (j == (count_width / 2) - 1)
        {
          cout << "|";
        }
      }
    }
    cout << endl;
  }
}

int main(int argc,char **argv)
{
  double *matrix;
  int mat_size;

  cin >> mat_size;
  int count_x = mat_size;
  int count_y = mat_size * 2;
  matrix = (double *)malloc(count_x * count_y * sizeof(double));

  for (int i = 0; i < mat_size; i++)
  {
    for (int j = 0; j < mat_size; j++)
    {
      cin >> *(matrix + i * count_y + j);
    }
  }

  for (int i = 0; i < mat_size; i++)
  {
    for (int j = mat_size; j < (mat_size) * 2; j++)
    {
      if (i + mat_size == j)
      {
        *(matrix + i * count_y + j) = 1;
      }
      else
      {
        *(matrix + i * count_y + j) = 0;
      }
    }
  }


  dim3 blockDim(16);
  dim3 gridDim((count_y + blockDim.x - 1) / blockDim.x);

  double *device_matrix_pointer;

  hipMalloc((void**)&device_matrix_pointer, count_x * count_y * sizeof(double));
  hipMemcpy(device_matrix_pointer, matrix,  count_x * count_y * sizeof(double), hipMemcpyHostToDevice);

  printf("grid.x %d grid.y %d grid.z %d\n",gridDim.x,gridDim.y,gridDim.z);
  printf("block.x %d block.y %d block.z %d\n",blockDim.x,blockDim.y,blockDim.z);

  for (int i = 0; i < count_x; i++) {
    calculateInverse<<<blockDim,gridDim>>>(device_matrix_pointer, count_y, count_x, i);
    hipDeviceSynchronize();
  }

  hipMemcpy(matrix, device_matrix_pointer, count_x * count_y * sizeof(double), hipMemcpyDeviceToHost);

  printMatrix(matrix, count_y, count_x, 0);

  hipFree(device_matrix_pointer);
  hipDeviceReset();
  return 0;
}