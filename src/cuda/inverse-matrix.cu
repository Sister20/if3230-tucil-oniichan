#include "hip/hip_runtime.h"
#include <stdio.h>
#include "inverse-matrix.cuh"

using namespace std;

__global__ void calculateInverse(double *matrix_pointer, int width, int height, int currRow)
{
  __shared__ double diagonal;
  diagonal = matrix_pointer[width * currRow + currRow];
  __syncthreads();

  // step1(matrix_pointer, width, height, currRow, diagonal);
  __syncthreads();
  // step2(matrix_pointer, width, height, currRow, diagonal);
}

__global__ void step1(double *matrix_pointer, int width, int height, int currRow)
{
  __shared__ double diagonal;
  diagonal = matrix_pointer[width * currRow + currRow];
  __syncthreads();

  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < width)
  {
    matrix_pointer[currRow * width + idx] = matrix_pointer[currRow * width + idx] / diagonal;
  }
  __syncthreads();
}

__global__ void step2(double *matrix_pointer, int width, int height, int currRow)
{
  int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
  __syncthreads();
  // int col_idx = blockIdx.y * blockDim.y + threadIdx.y;

  // if (row_idx != currRow && row_idx < height && col_idx < width && col_idx >= currRow) {
  //     int pivotAdjIdx = row_idx * width + currRow;
  //     double multiplier = matrix_pointer[pivotAdjIdx];

  //     int idx = row_idx * width + col_idx;
  //     int pivotIdx = currRow * width + col_idx;

  //     double before = matrix_pointer[idx];
  //     double pivotElmt = matrix_pointer[pivotIdx];

  //     matrix_pointer[idx] = before - multiplier * pivotElmt;

  //     // double after = matrix_pointer[idx];

  //     // printf("iteration: %d, Col: %d, Row: %d, Multiplier: %.15f, pivotElmt %.15f, before %.15f, after %.15f, height %d width %d\n", currRow, col_idx, row_idx, multiplier, pivotElmt, before, after, height, width);
  // }

  if (row_idx != currRow && row_idx < height)
  {
    // printf("Row: %d\n", row_idx);
    int pivotAdjIdx = row_idx * width + currRow;
    double multiplier = matrix_pointer[pivotAdjIdx];
    for (int col = 0; col < width; col++)
    {
      int idx = row_idx * width + col;
      int pivotIdx = currRow * width + col;

      double before = matrix_pointer[idx];
      double pivotElmt = matrix_pointer[pivotIdx];

      matrix_pointer[idx] = before - multiplier * pivotElmt;

      double after = matrix_pointer[idx];

      // printf("iteration: %d, Col: %d, Row: %d, Multiplier: %.15f, pivotElmt %.15f, before %.15f, after %.15f, height %d width %d\n", currRow, col, row_idx, multiplier, pivotElmt, before, after, height, width);
    }
  }
  __syncthreads();
}