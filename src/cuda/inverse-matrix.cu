#include "hip/hip_runtime.h"
#include <stdio.h>
#include "inverse-matrix.cuh"

using namespace std;

__global__ void calculateInverse(double* matrix_pointer, int width, int height, int currRow)
{
  __shared__ double diagonal;
  diagonal = matrix_pointer[width * currRow + currRow];
  __syncthreads();

  step1(matrix_pointer, width, height, currRow, diagonal);
  __syncthreads();
  step2(matrix_pointer, width, height, currRow, diagonal);
}

__device__ void step1(double* matrix_pointer, int width, int height, int currRow, double diagonal) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < width) {
    matrix_pointer[currRow * width + idx] =  matrix_pointer[currRow * width + idx] / diagonal;
  }

}

__device__ void step2(double* matrix_pointer, int width, int height, int currRow, double diagonal) {
  int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int col_idx = blockIdx.y * blockDim.y + threadIdx.y;

  if (row_idx != currRow && row_idx < height && col_idx < width) {
//     // printf("Row: %d\n", row_idx);
//     // int pivotAdjIdx = row_idx * width + currRow;
//     // double multiplier = matrix_pointer[pivotAdjIdx];
//     // for (int col = 0; col < width; col++) {
//     //   int idx = row_idx * width + col;
//     //   int pivotIdx = currRow * width + col;
//     //   matrix_pointer[idx] = matrix_pointer[idx] - multiplier * matrix_pointer[pivotIdx];
//     // }
    double multiplier = matrix_pointer[row_idx * width + currRow];
    printf("Multiplier: %.15f\n", multiplier);
//     int idx = row_idx * width + col_idx;
//     matrix_pointer[idx] = matrix_pointer[idx] - multiplier * matrix_pointer[currRow * width + col_idx];
  }

}